#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h> //#define assert
#include <hip/hip_runtime.h>
#include "snp_model.hpp"
#include "error_check.hpp"

using namespace std;

/** Allocation */
SNP_model::SNP_model(uint n, uint m)
{
    this->step = 0;
    this->cpu_updated = 1;
    this->gpu_updated = 0;
    // allocation in CPU
    this->n = n;  // number of neurons
    this->m = m;  // number of rules
    this->conf_vector     = (uint*) malloc(sizeof(uint)*n); // configuration vector (only one, we simulate just a computation)
    this->delays_vector = (uint*) malloc(sizeof(uint)*n); 
    this->rule_index      = (int*)   malloc(sizeof(int)*(n+1)); // indeces of rules inside neuron (start index per neuron)
    this->rules.Ei        = (uint*)  malloc(sizeof(uint)*m); // Regular expression Ei of a rule
    this->rules.En        = (uint*)  malloc(sizeof(uint)*m); // Regular expression En of a rule
    this->rules.c         = (uint*)  malloc(sizeof(uint)*m); // LHS of rule
    this->rules.p         = (uint*)  malloc(sizeof(uint)*m); // RHS of rule
    this->rules.d         = (uint*)  malloc(sizeof(uint)*m); // RHS of rule
    this->rules.nid       = (uint*)   malloc(sizeof(uint)*(m)); // Index of the neuron where the rule is
    this->calc_next_trans = (bool*) malloc(sizeof(bool));

    // allocation in GPU
    hipMalloc(&this->d_conf_vector,   sizeof(uint)*n);
    hipMalloc(&this->d_delays_vector,   sizeof(uint)*n);
    hipMalloc(&this->d_rule_index,    sizeof(int)*(n+1));
    hipMalloc(&this->d_rules.Ei,      sizeof(uint)*m);
    hipMalloc(&this->d_rules.En,      sizeof(uint)*m);
    hipMalloc(&this->d_rules.c,       sizeof(uint)*m);
    hipMalloc(&this->d_rules.p,       sizeof(uint)*m);
    hipMalloc(&this->d_rules.d,       sizeof(uint)*m);
    hipMalloc(&this->d_rules.nid,     sizeof(uint)*m);
    hipMalloc(&this->d_calc_next_trans, sizeof(bool));

    // initialization (only in CPU, having updated version)
    memset(this->conf_vector,   0,  sizeof(uint)*n);
    memset(this->delays_vector,   0,  sizeof(uint)*n);
    memset(this->rule_index,    -1,  sizeof(int)*(n+1));
    this->rule_index[0]=0;
    memset(this->rules.Ei,      0,  sizeof(uint)*m);
    memset(this->rules.En,      0,  sizeof(uint)*m);
    memset(this->rules.c,       0,  sizeof(uint)*m);
    memset(this->rules.p,       0,  sizeof(uint)*m);
    memset(this->rules.d,       0,  sizeof(uint)*n);
    memset(this->rules.nid,     0,  sizeof(uint)*(m));
   
    // memory consistency, who has the updated copy?
    gpu_updated = false; cpu_updated = true;
    done_rules = false;
}

/** Free mem */
SNP_model::~SNP_model()
{
    free(this->conf_vector);
    free(this->spiking_vector);
    free(this->trans_matrix);
    free(this->rule_index);
    free(this->rules.Ei);
    free(this->rules.En);
    free(this->rules.c);
    free(this->rules.p);
    free(this->rules.d);
    free(this->rules.nid);
    free(this->calc_next_trans);
    free(this->delays_vector);

    hipFree(this->d_conf_vector);
    hipFree(this->d_spiking_vector);
    hipFree(this->d_trans_matrix);
    hipFree(this->d_rule_index);
    hipFree(this->d_rules.Ei);
    hipFree(this->d_rules.En);
    hipFree(this->d_rules.c);
    hipFree(this->d_rules.p);
    hipFree(this->d_rules.d);
    hipFree(this->d_rules.nid);
    hipFree(this->d_calc_next_trans);
    hipFree(this->d_delays_vector);
}

void SNP_model::print_conf_vector (){
    //////////////////////////////////////////////////////
    assert(gpu_updated || cpu_updated);
    if (!cpu_updated) load_to_cpu();
    //////////////////////////////////////////////////////
    
    printf("Configuration vector\n");
    for(int i=0; i<n; i++){
        printf("%d ",conf_vector[i]);
    }
    printf("\n");
}

void SNP_model::set_snpconfig (int verbosity_lv, int repetitions, char *outfile){
    this->verbosity_lv = verbosity_lv;
    this->repetitions = repetitions;
    this->outfile = outfile;
}

void SNP_model::set_spikes (uint nid, uint s)
{
    //////////////////////////////////////////////////////
    assert(nid < n);
    // check memory consistency, who has the updated copy?
    assert(gpu_updated || cpu_updated);
    if (gpu_updated && !cpu_updated) load_to_cpu();
    gpu_updated = false;
    //////////////////////////////////////////////////////

    conf_vector[nid] = s;    
}

uint SNP_model::get_spikes (uint nid)
{
    //////////////////////////////////////////////////////
    assert(nid < n);
    // check memory consistency, who has the updated copy?
    assert(gpu_updated || cpu_updated);
    if (gpu_updated && !cpu_updated) load_to_cpu();
    //////////////////////////////////////////////////////

    return conf_vector[nid];
}

/** Add a rule to neuron nid, regular expression defined by e_n and e_i, and a^c -> a^p.
    Must be called sorted by neuron */
void SNP_model::add_rule (uint nid, uint e_n, uint e_i, uint c, uint p, uint d) 
{
    //////////////////////////////////////////////////////
    assert(nid < n);
    assert(!done_rules);
    // check memory consistency, who has the updated copy?
    assert(gpu_updated || cpu_updated);
    gpu_updated = false; cpu_updated = true;
    //////////////////////////////////////////////////////

    if (rule_index[nid+1] == -1) // first rule in neuron
        rule_index[nid+1] = rule_index[nid] + 1; 
    else   // keep accumulation
        rule_index[nid+1] = rule_index[nid+1] + 1;

    uint rid = rule_index[nid+1]-1;

    rules.Ei[rid] = e_i;
    rules.En[rid] = e_n;
    rules.c[rid]  = c;
    rules.p[rid]  = p;
    rules.d[rid]  = d;
    rules.nid[rid]= nid;
}

/** Add synapse from neuron i to j. 
    Must be called after adding all rules */
void SNP_model::add_synapse (uint i, uint j) 
{
    //////////////////////////////////////////////////////
    // ensure parameters within limits
    assert(i < n && j < n);
    // ensure all rules have been introduced already
    // assert(rule_index[n]==m);
    // SNP does not allow self-synapses
    assert(i!=j);
    done_rules = true; // from now on, no more rules can be added
    // check memory consistency, who has the updated copy?
    assert(gpu_updated || cpu_updated);
    gpu_updated = false; cpu_updated = true;
    //////////////////////////////////////////////////////

    include_synapse(i,j);
}
__global__ void k_print_conf_v(uint *conf_v, int n){
    printf("Configuration vector(gpu memory)\n");
    for(int i=0; i<n; i++){
        printf("%d ",conf_v[i]);
    }
    printf("\n");
}



bool SNP_model::transition_step ()
{
    //////////////////////////////////////////////////////
    // check memory consistency, who has the updated copy?
    assert(gpu_updated || cpu_updated);
    if (!gpu_updated) load_to_gpu();
    //////////////////////////////////////////////////////
    if(step==0 && verbosity_lv >= 3){
        print_transition_matrix();
        print_conf_vector();
    }
    cpu_updated = false;

    bool calc_next = false;

    calc_spiking_vector();
    if(verbosity_lv >= 3){
        print_spiking_vector();
        print_delays_vector();
    }
    calc_next = check_next_trans();
    
    if(calc_next){
        if(verbosity_lv >= 2){
            printf("\n\nstep #%d",step);
            printf("\n---------------------------------------\n");
        }

        calc_transition();
        if(verbosity_lv >= 2){
            print_conf_vector();
        }
        step++;
        return calc_next;
    }
    
    if(verbosity_lv==1){
        printf("\nstep #%d\n",step);
        print_conf_vector();
    }

    return calc_next; 
}

void SNP_model::load_to_gpu () 
{
    //////////////////////////////////////////////////////
    // check memory consistency, who has the updated copy?
    assert(gpu_updated || cpu_updated);
    if (gpu_updated) return;
    gpu_updated = true;
    //////////////////////////////////////////////////////

    hipMemcpy(d_conf_vector,   conf_vector,    sizeof(uint)*n,   hipMemcpyHostToDevice);
    hipMemcpy(d_rule_index,    rule_index,     sizeof(uint)*(n+1), hipMemcpyHostToDevice);
    hipMemcpy(d_rules.Ei,      rules.Ei,       sizeof(uint)*m,    hipMemcpyHostToDevice);
    hipMemcpy(d_rules.En,      rules.En,       sizeof(uint)*m,    hipMemcpyHostToDevice);
    hipMemcpy(d_rules.c,       rules.c,        sizeof(uint)*m,    hipMemcpyHostToDevice);
    hipMemcpy(d_rules.p,       rules.p,        sizeof(uint)*m,    hipMemcpyHostToDevice);
    hipMemcpy(d_rules.d,       rules.d,        sizeof(uint)*m,    hipMemcpyHostToDevice);
    hipMemcpy(d_rules.nid,     rules.nid,      sizeof(uint)*m,     hipMemcpyHostToDevice);

    load_spiking_vector();
    load_transition_matrix();
}

void SNP_model::load_to_cpu ()
{
    //////////////////////////////////////////////////////
    // check memory consistency, who has the updated copy?
    assert(gpu_updated || cpu_updated);
    if (cpu_updated) return;
    cpu_updated = true;
    //////////////////////////////////////////////////////

    hipMemcpy(conf_vector, d_conf_vector, sizeof(uint)*n, hipMemcpyDeviceToHost);
}



