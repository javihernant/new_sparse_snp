#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <snp_static.hpp>
#include "error_check.hpp"

using namespace std;

/** Allocation */
SNP_static_sparse::SNP_static_sparse(uint n, uint m) : SNP_model(n,m)
{
    // n is num of rows, m is num of colums. 
    // done by subclasses
    this->trans_matrix    = (int*)  malloc(sizeof(int)*n*m);
    this->spiking_vector  = (int*) malloc(sizeof(int)*m); // spiking vector

    memset(this->trans_matrix,0,sizeof(int)*n*m);

    cuda_check(hipMalloc(&this->d_trans_matrix,  sizeof(int)*n*m));
    cuda_check(hipMalloc(&this->d_spiking_vector,sizeof(int)*m));

    cuda_check(hipMemset(this->d_spiking_vector, -1, sizeof(int)*m));

}

SNP_static_ell::SNP_static_ell(uint n, uint m) : SNP_model(n,m)
{
    //Allocate cpu variables
    this->trans_matrix    = (int*)  malloc(sizeof(int)*n*m*2);
    this -> spiking_vector = (int*) malloc(sizeof(int)*m);
    this->z_vector    = (int*) malloc(sizeof(int)*m);
    
    memset(this->trans_matrix,-1,sizeof(int)*n*m*2);
    memset(this->z_vector,0,sizeof(int)*m);
    this->z = 0;

    //Allocate device variables
    cuda_check(hipMalloc((&this->d_spiking_vector),  sizeof(int)*m));

    cuda_check(hipMemset(this->d_spiking_vector, -1, sizeof(int)*m));
    //trans_matrix allocated when z is known
}

SNP_static_optimized::SNP_static_optimized(uint n, uint m) : SNP_model(n,m)
{
    //Allocate cpu variables
    this->trans_matrix    = (int*)  malloc(sizeof(int)*n*n);
    this -> spiking_vector = (int*) malloc(sizeof(int)*n);
    this->z_vector    = (int*) malloc(sizeof(int)*n);
    
    memset(this->trans_matrix,-1,sizeof(int)*n*n);
    memset(this->z_vector,0,sizeof(int)*n);
    this->z=0;

    //Allocate device variables
    cuda_check(hipMalloc((&this->d_spiking_vector),  sizeof(int)*n));
    //d_trans_matrix allocated when z is known

    cuda_check(hipMemset(this->d_spiking_vector, -1, sizeof(int)*n));
}

/** Free mem */
SNP_static_ell::~SNP_static_ell()
{
    free(this->z_vector);
}

SNP_static_optimized::~SNP_static_optimized()
{
    free(this->z_vector);
}

void SNP_static_sparse::print_transition_matrix(){
    printf("Transition matrix\n");

    for(int i=0; i<m; i++){
        for(int j=0; j<n; j++){
            printf("%d ",trans_matrix[i*n + j]);
        }  
        printf("\n");
    }
    printf("\n");
}

void SNP_static_ell::print_transition_matrix(){
    assert(z > 0);
    printf("Transition matrix\n");

    for(int i=0; i<z; i++){
        for(int j=0; j<m; j++){
            int idx = (i*m*2 + j*2);
            printf("(%d, %d)",trans_matrix[idx], trans_matrix[idx+1]);
        }  
        printf("\n");
    }
    printf("\n");
}

void SNP_static_optimized::print_transition_matrix(){
    assert(z > 0);
    printf("Transition matrix\n");

    for(int i=0; i<z; i++){
        for(int j=0; j<n; j++){
            int idx = (i*n + j);
            printf("%d ",trans_matrix[idx]);
        }  
        printf("\n");
    }
    printf("\n");
}

__global__ void k_print_trans_mx_sparse(int *mx, int n, int m){
    printf("Transition matrix(gpu memory)\n");

    for(int i=0; i<m; i++){
        for(int j=0; j<n; j++){
            printf("%d ",mx[i*n + j]);
        }  
        printf("\n");
    }
    printf("\n");
}


__global__ void k_print_spk_v_generic(int *spkv, int m){
    printf("Spiking vector\n");
    for(int i=0; i<m; i++){
        printf("%d ", spkv[i]);
    }
    printf("\n");
}

__global__ void k_print_dys_v_generic(uint *dys, int n){
    printf("Delays vector\n");
    for(int i=0; i<n; i++){
        printf("%d ", dys[i]);
    }
    printf("\n");
}

void SNP_static_sparse::print_spiking_vector(){
    //print from gpu
    // k_print_trans_mx_sparse<<<1,1>>>(d_trans_matrix, n, m);
    k_print_spk_v_generic<<<1,1>>>(d_spiking_vector, m);
    hipDeviceSynchronize();
}

void SNP_static_ell::print_spiking_vector(){
    k_print_spk_v_generic<<<1,1>>>(d_spiking_vector, m);
    hipDeviceSynchronize();
}

void SNP_static_optimized::print_spiking_vector(){
    k_print_spk_v_generic<<<1,1>>>(d_spiking_vector, n);
    hipDeviceSynchronize();
}

void SNP_static_sparse::print_delays_vector(){
    k_print_dys_v_generic<<<1,1>>>(d_delays_vector, n);
    hipDeviceSynchronize();
}

void SNP_static_ell::print_delays_vector(){
    k_print_dys_v_generic<<<1,1>>>(d_delays_vector, n);
    hipDeviceSynchronize();
}

void SNP_static_optimized::print_delays_vector(){
    k_print_dys_v_generic<<<1,1>>>(d_delays_vector, n);
    hipDeviceSynchronize();
}

__global__ void kalc_spiking_vector_generic(int* spiking_vector, uint* conf_vector, int* rule_index, uint* rei, uint* ren, uint* rc, uint* rd, uint* delays_vector, uint n)
{
    uint nid = threadIdx.x+blockIdx.x*blockDim.x;

    if (nid<n && delays_vector[nid]==0) {
        for (int r=rule_index[nid]; r<rule_index[nid+1]; r++){
            uchar e_i = rei[r];
            uchar e_n = ren[r];
            int x = conf_vector[nid];
            if ((int) (e_i&(x==e_n)) || ((1-e_i)&(x>=e_n))) {   
                conf_vector[nid]-=rc[r];
                
                /*handle situation where rule was previously selected, neuron had to wait d steps, but right when it is 
                about to be fired same rule is selected, losing the action of the first time it was selected.
                When this situation appears in rule r, spiking vector[r] is incremented by one, meaning one instance of rule 
                r has already completed delay time and is waiting to be fired. For performance purposes, all instances 
                of rules in this situation will be applied together when its delay counter gets to 0 */

                if(spiking_vector[r] > 0 && delays_vector[nid] == 0){
                    spiking_vector[r]+=1;
                }else{
                    spiking_vector[r] = 1;
                }
                
                delays_vector[nid] = rd[r];
                break;
            }
        }
    }
}

__global__ void kalc_spiking_vector_for_optimized(int* spiking_vector, uint* conf_vector, int* rule_index, uint* rei, uint* ren, uint* rc, uint* rd, uint* delays_vector, uint n)
{
    uint nid = threadIdx.x+blockIdx.x*blockDim.x;
    if (nid<n && delays_vector[nid]==0) {
        //vector<int> active_rule_idxs_ni;
        for (int r=rule_index[nid]; r<rule_index[nid+1]; r++){
            uchar i = rei[r];
            uchar n = ren[r];
            int x = conf_vector[nid];
            if (((int) (i&(x==n)) || ((1-i)&(x>=n)))){
                //active_ridx.push_back(r);
                delays_vector[nid] = rd[r];
                conf_vector[nid]-=rc[r];
                spiking_vector[nid] = r;
                break;
            }
        }
    }
}

void SNP_static_sparse::calc_spiking_vector() 
{
    //////////////////////////////////////////////////////
    cpu_updated = false;
    //////////////////////////////////////////////////////

    uint bs = 256;
    uint gs = (m+255)/256;
    
    kalc_spiking_vector_generic<<<gs,bs>>>(d_spiking_vector, d_conf_vector, d_rule_index, d_rules.Ei, d_rules.En, d_rules.c, d_rules.d, d_delays_vector, n);
    cuda_check(hipGetLastError());
    hipDeviceSynchronize();
}

void SNP_static_ell::calc_spiking_vector() 
{
    uint bs = 256;
    uint gs = (m+255)/256;
    
    kalc_spiking_vector_generic<<<gs,bs>>>(d_spiking_vector, d_conf_vector, d_rule_index, d_rules.Ei, d_rules.En, d_rules.c, d_rules.d, d_delays_vector, n);
    cuda_check(hipGetLastError());
    hipDeviceSynchronize();
}

void SNP_static_optimized::calc_spiking_vector() 
{
    uint bs = 256;
    uint gs = (m+255)/256;
    kalc_spiking_vector_for_optimized<<<gs,bs>>>(d_spiking_vector, d_conf_vector, d_rule_index, d_rules.Ei, d_rules.En, d_rules.c, d_rules.d, d_delays_vector, n);
    cuda_check(hipGetLastError());
    hipDeviceSynchronize();
}

void SNP_static_sparse::include_synapse(uint i, uint j)
{
    for (int r = rule_index[i]; r < rule_index[i+1]; r++) {
        trans_matrix[r*n+i] = 0;  
        trans_matrix[r*n+j] = rules.p[r];
    }
}

void SNP_static_ell::include_synapse(uint i, uint j)
{
    for (int r = rule_index[i]; r < rule_index[i+1]; r++) {
        //forgeting rules are not stored in trans_mx. 
        if(rules.p[r]>0){
            trans_matrix[z_vector[r]*m*2+r*2] = j;
            trans_matrix[(z_vector[r]*m*2+r*2)+1] = rules.p[r];
            z_vector[r]++;
        }
    }
}

void SNP_static_optimized::include_synapse(uint i, uint j)
{
    trans_matrix[z_vector[i]*n+i] = j;
    z_vector[i]++;
}


void SNP_static_sparse::load_transition_matrix () 
{
    cuda_check(hipMemcpy(d_trans_matrix, trans_matrix, sizeof(int)*n*m, hipMemcpyHostToDevice));
}

void SNP_static_ell::load_transition_matrix () 
{
    for(int r=0; r<m; r++){
        int aux_z=z_vector[r];
        if(aux_z>z){
            z=aux_z;
        }
    }
    assert(z>0);

    cuda_check(hipMalloc((&this->d_trans_matrix),  sizeof(int)*z*m*2));
    cuda_check(hipMemcpy(d_trans_matrix, trans_matrix, sizeof(int)*z*m*2, hipMemcpyHostToDevice));
}

void SNP_static_optimized::load_transition_matrix (){

    for(int i=0; i<n; i++){
        int z_aux = z_vector[i];
        if(z_aux>z){
            z = z_aux;    
        }
    }

    // this-> trans_matrix = (int *) realloc(this->trans_matrix,sizeof(int)*n*z);
    cuda_check(hipMalloc((&this->d_trans_matrix),  sizeof(int)*n*z));
    hipMemcpy(d_trans_matrix,  trans_matrix,   sizeof(int)*n*z,  hipMemcpyHostToDevice);
}

__global__ void kalc_transition_sparse(int* spiking_vector, int* trans_matrix, uint* conf_vector,uint * delays_vector, uint * rnid , uint n, uint m){
    int nid = threadIdx.x+blockIdx.x*blockDim.x;
    if (nid<n && delays_vector[nid]==0){
        for (int r=0; r<m; r++){
            
            if(spiking_vector[r] != -1 && delays_vector[rnid[r]]==0){
                conf_vector[nid] += spiking_vector[r] * trans_matrix[r*n+nid];
            }
            
            __syncthreads(); //disable rule when all threads have finished processing row (using only one thread)
            if(nid==0 && spiking_vector[r] != -1 && delays_vector[rnid[r]]==0){
                spiking_vector[r] = -1; 
            }
            
        }
    }
}

__global__ void kalc_transition_ell(int* spiking_vector, int* trans_matrix, uint* conf_vector,uint * delays_vector, uint * rnid , uint z, uint m){
    int rid = threadIdx.x+blockIdx.x*blockDim.x;
    if (rid<m && spiking_vector[rid]>0 && delays_vector[rnid[rid]]==0){
        for(int i=0; i<z; i++){
            int neuron = trans_matrix[m*2*i+rid*2];
            int value = trans_matrix[m*2*i+rid*2+1];
            if(neuron==-1 && value==-1){
                break;
            }
            if(delays_vector[neuron]==0){
                //mult value times number of followed activation of a rule (ie. spiking_vector[rid])
                atomicAdd((uint *)&conf_vector[neuron], (uint)value*spiking_vector[rid]);
            }        
        }
        spiking_vector[rid] = -1;
    }
}

__global__ void kalc_transition_optimized(int* spiking_vector, int* trans_matrix, uint* conf_vector, uint* delays_vector, uint* rc, uint* rp, int z, uint n){
    int nid = threadIdx.x+blockIdx.x*blockDim.x;

    if(nid<n && spiking_vector[nid]>=0 && delays_vector[nid]==0){
        int rid = spiking_vector[nid];
        int p = rp[rid];
        // printf("nid:%d, rid:%d, c:%d, p:%d\n", nid, rid, c, p);

        for(int j=0; j<z; j++){
            int n_j = trans_matrix[j*n+nid]; //nid is connected to n_j. 

            if(n_j >= 0){
                if(delays_vector[n_j]>0) break;
                atomicAdd((int *) &conf_vector[n_j], p);
            }else{
                //if padded value (-1)
                break;
            }
        }
        spiking_vector[nid]= -1;
    }
}

__global__ void update_delays_vector_generic(uint *delays_vector, uint n){
    
    int nid = threadIdx.x+blockIdx.x*blockDim.x;
    if(nid<n && delays_vector[nid]>0){
        delays_vector[nid]--;
    }
}

__global__ void k_check_next_trans(bool *calc_nxt, int* spkv, int spkv_size, uint * delays, int neurons){
    calc_nxt[0] = false;
    
    for(int i=0; i<spkv_size; i++){
        if(spkv[i] !=-1){
            calc_nxt[0] = true;
            break;   
        }
    }

    if(!calc_nxt[0]){   
        for(int i=0; i<neurons; i++){
            if(delays[i] > 0){
                calc_nxt[0] = true;
                break;
            }
        }
    }
}

bool SNP_static_sparse::check_next_trans(){
    k_check_next_trans<<<1,1>>>(d_calc_next_trans, d_spiking_vector, m, d_delays_vector, n);
    hipDeviceSynchronize();
    cuda_check(hipMemcpy(calc_next_trans, d_calc_next_trans, sizeof(bool),hipMemcpyDeviceToHost));
    // printf("calc_next:%d",calc_next_trans[0]);
    return calc_next_trans[0];
}

bool SNP_static_ell::check_next_trans(){
    k_check_next_trans<<<1,1>>>(d_calc_next_trans, d_spiking_vector, m, d_delays_vector, n);
    hipDeviceSynchronize();
    cuda_check(hipMemcpy(calc_next_trans, d_calc_next_trans, sizeof(bool),hipMemcpyDeviceToHost));
    return calc_next_trans[0];
}

bool SNP_static_optimized::check_next_trans(){
    k_check_next_trans<<<1,1>>>(d_calc_next_trans, d_spiking_vector, n, d_delays_vector, n);
    hipDeviceSynchronize();
    cuda_check(hipMemcpy(calc_next_trans, d_calc_next_trans, sizeof(bool),hipMemcpyDeviceToHost));
    return calc_next_trans[0];
}

void SNP_static_sparse::calc_transition()
{
    //////////////////////////////////////////////////////
    cpu_updated = false;
    //////////////////////////////////////////////////////

    kalc_transition_sparse<<<n+255,256>>>(d_spiking_vector,d_trans_matrix, d_conf_vector, d_delays_vector, d_rules.nid,n,m);
    cuda_check(hipGetLastError());
    update_delays_vector_generic<<<n+255,256>>>(d_delays_vector, n);
    cuda_check(hipGetLastError());
    hipDeviceSynchronize();
}

void SNP_static_ell::calc_transition()
{
    //////////////////////////////////////////////////////
    cpu_updated = false;
    //////////////////////////////////////////////////////

    kalc_transition_ell<<<n+255,256>>>(d_spiking_vector,d_trans_matrix, d_conf_vector, d_delays_vector, d_rules.nid,z,m);
    cuda_check(hipGetLastError());
    update_delays_vector_generic<<<n+255,256>>>(d_delays_vector, n);
    cuda_check(hipGetLastError());
    hipDeviceSynchronize();
}

void SNP_static_optimized::calc_transition()
{
    //////////////////////////////////////////////////////
    cpu_updated = false;
    //////////////////////////////////////////////////////

    kalc_transition_optimized<<<n+255,256>>>(d_spiking_vector,d_trans_matrix, d_conf_vector, d_delays_vector, d_rules.c, d_rules.p, z,n);
    cuda_check(hipGetLastError());
    update_delays_vector_generic<<<n+255,256>>>(d_delays_vector, n);
    cuda_check(hipGetLastError());
    hipDeviceSynchronize();
}